#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here


//***********************************************


__global__ void isAlive(int* match, int* isTankAlive, int* dHealth, int T) {
    if (T >= threadIdx.x) {
        int id_ = threadIdx.x;
        if (dHealth[id_] <= 0)
        {
            match++;
            atomicAdd(isTankAlive, -1);
        }
    }
}


__global__ void simulate_round(int M, int N, int T, int round, int* dHealth, int* dHealth_copy, int* dScore, int* dXC, int* dYC) {

    long long idx_source = 1LL * dYC[blockIdx.x] * N + dXC[blockIdx.x];
    long long idx_tar = 1LL * dYC[(blockIdx.x + round) % T] * N + dXC[(blockIdx.x + round) % T];
    long long idx_mid = 1LL * dYC[threadIdx.x] * N + dXC[threadIdx.x];

    __shared__ long long min_dis;
    min_dis = LLONG_MAX;

    int x1 = dXC[blockIdx.x];
    int x2 = dXC[(blockIdx.x + round) % T];
    int x3 = dXC[threadIdx.x];

    int y1 = dYC[blockIdx.x];
    int y2 = dYC[(blockIdx.x + round) % T];
    int y3 = dYC[threadIdx.x];


    bool slope = false;
    if ((1LL * (y1 - y2) * (x1 - x3)) == (1LL * (y1 - y3) * (x1 - x2)))
        slope = true;


    bool common_area = false;
    if ((idx_tar > idx_source) == false && ((idx_mid > idx_source)) == false)
        common_area = true;
    else if ((idx_tar > idx_source) == true && ((idx_mid > idx_source)) == true)
        common_area = true;

    __syncthreads();


    long long middle_dis = abs(idx_source - idx_mid);


    if (blockIdx.x != threadIdx.x)
    {
        if (dHealth[blockIdx.x] > 0)
        {
            if (slope && common_area)
            {
                if (dHealth[threadIdx.x] > 0)
                    atomicMin(&min_dis, middle_dis);
            }
        }
    }__syncthreads();




    if (blockIdx.x != threadIdx.x)
    {
        if (dHealth[blockIdx.x] > 0)
        {
            if (slope && common_area)
            {
                if (dHealth[threadIdx.x] > 0)
                {
                    if (min_dis == middle_dis)
                    {
                        atomicAdd(dHealth_copy + threadIdx.x, -1);
                    }
                    int iid_ = blockIdx.x;
                    if (min_dis == middle_dis) {
                        dScore[iid_]++;
                    }

                }
            }
        }
    }


}


__global__ void init_HP(int* dHealth, int* dHealth_cp, int H, int T) {

    int id = threadIdx.x;
    if (id < T)
    {
        dHealth[threadIdx.x] = H;
    }
    if (T >= id)
    {
        dHealth_cp[id] = H;
    }
}



int main(int argc, char** argv)
{
    // Variable declarations
    int M, N, T, H, * xcoord, * ycoord, * score;


    FILE* inputfilepointer;

    //File Opening for read
    char* inputfilename = argv[1];
    inputfilepointer = fopen(inputfilename, "r");

    if (inputfilepointer == NULL) {
        printf("input.txt file failed to open.");
        return 0;
    }

    fscanf(inputfilepointer, "%d", &M);
    fscanf(inputfilepointer, "%d", &N);
    fscanf(inputfilepointer, "%d", &T); // T is number of Tanks
    fscanf(inputfilepointer, "%d", &H); // H is the starting Health point of each Tank

    // Allocate memory on CPU
    xcoord = (int*)malloc(T * sizeof(int));  // X coordinate of each tank
    ycoord = (int*)malloc(T * sizeof(int));  // Y coordinate of each tank
    score = (int*)malloc(T * sizeof(int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for (int i = 0; i < T; i++)
    {
        fscanf(inputfilepointer, "%d", &xcoord[i]);
        fscanf(inputfilepointer, "%d", &ycoord[i]);
    }


    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************


    //create array for health


    int* dScore;
    int* current_alive;
    hipMalloc(&dScore, (T * sizeof(int)));
    hipMemset(dScore, 0, T * sizeof(int));

    int* dHealth;
    hipMalloc(&dHealth, (T * sizeof(int)));

    int* match;
    hipMalloc(&match, sizeof(int));


    int* dXC;
    hipMalloc(&dXC, (T * sizeof(int)));
    hipMemcpy(dXC, xcoord, sizeof(int) * T, hipMemcpyHostToDevice);

    int* dYC;
    hipMalloc(&dYC, (sizeof(int) * T));
    hipMemcpy(dYC, ycoord, sizeof(int) * T, hipMemcpyHostToDevice);

    int* dHealth_copy;
    current_alive = (int*)malloc(sizeof(int));
    hipMalloc(&dHealth_copy, (T * sizeof(int)));

    int* dcurrent_alive;
    hipMalloc(&dcurrent_alive, sizeof(int));


    *current_alive = T;

    init_HP << <1, T >> > (dHealth, dHealth_copy, H, T);

    int round = 1;
    while (1)
    {
        if (*current_alive <= 1)
            break;
        else {
            hipMemcpy(dcurrent_alive, &T, sizeof(int), hipMemcpyHostToDevice);
            if (round % T != 0) {
                simulate_round << <T, T >> > (M, N, T, round, dHealth, dHealth_copy, dScore, dXC, dYC);
                int size_ = T * sizeof(int);
                match++;
                hipMemcpy(dHealth, dHealth_copy, size_, hipMemcpyDeviceToDevice);
                round++;
                isAlive << <1, T >> > (match, dcurrent_alive, dHealth, T);
                size_ /= T;
                hipMemcpy(current_alive, dcurrent_alive, size_, hipMemcpyDeviceToHost);
            }
            else if (round % T == 0)
                round++;
        }
    }



    hipFree(dHealth);
    hipFree(dXC);
    hipMemcpy(score, dScore, sizeof(int) * T, hipMemcpyDeviceToHost);
    hipFree(dScore);
    hipFree(dHealth_copy);
    hipFree(dYC);


    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end - start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char* outputfilename = argv[2];
    char* exectimefilename = argv[3];
    FILE* outputfilepointer;
    outputfilepointer = fopen(outputfilename, "w");

    for (int i = 0; i < T; i++)
    {
        fprintf(outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename, "w");
    fprintf(outputfilepointer, "%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;

}
